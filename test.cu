#include "process.cu"

#include <iostream>

using namespace std;


int main() {

    int size = 8*1024;
    double *A = (double *)malloc(size*sizeof(double));
    double *Z = (double *)malloc(size*sizeof(double));
    for (int i = 0; i < size; i++) {
        // cout << i << endl;
        A[i] = 0;
        Z[i] = 0;
    }

    _cudaGrayScaleImage(A, 8, 1024, Z, 8, 1024, 3);
    return 0;
}