#include "hip/hip_runtime.h"
//  process.cpp
//  https://www.geeksforgeeks.org/how-to-call-c-c-from-python/

// Cuda Downloads
// https://docs.nvidia.com/cuda/cuda-toolkit-release-notes/index.html

#include <iostream>

#include "utils.cu"


__global__ void cudaGrayScale(double *y, int y_h, int y_w, double *x, int x_h, int x_w, int x_d, int threads) {

    int id = blockIdx.x * blockDim.x + threadIdx.x;
    int blockStart = blockIdx.x * blockDim.x;

    if (id == 0) {
        printf("Helllllloooooo");
    }

    int start;
    int stop;
    getConstraints(y_h, y_w, id, threads, blockIdx.x, blockStart,  blockDim.x, &start, &stop);
    printf("ID: %d, blockStart: %d, Start: %d, Stop: %d\n", id, blockStart, start, stop);

    // 0.2989 * r + 0.5870 * g + 0.1140 * b

    y[0] = 255;

}

__global__ void kernalTest(double *A) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    // if (id < 8)
    A[0] = id;
}

bool _cudaGrayScaleImage(double *y, int y_h, int y_w, double *x, int x_h, int x_w, int x_d) {


    int blocks = 8;  // 65535   thread.x, thread.y
    int threadsPerBlock = 1024;
    int totalThreadCount = blocks * threadsPerBlock; // 8196

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    int y_size = y_h * y_w * sizeof(double);
    int x_size = x_h * x_w * x_d * sizeof(double);

    // copying matrix into GPU
    double *cuda_y, *cuda_x;
    hipMalloc(&cuda_y, y_size);
    hipMalloc(&cuda_x, x_size);
    hipMemcpy(cuda_y, y, y_size, hipMemcpyHostToDevice);
    hipMemcpy(cuda_x, x, x_size, hipMemcpyHostToDevice);
    // printAll("y:", y_h, y_w, y);
    // printAll("x:", x_h, x_w, x);

    hipEventRecord(start);

    // Execute Kernal
    // cudaGrayScale<<< blocks, threadsPerBlock >>>(y, y_h, y_w, x, x_h, x_w, x_d, totalThreadCount);
    kernalTest<<< blocks, threadsPerBlock >>>(y);

    hipDeviceSynchronize();
    hipEventRecord(stop);


    // All Done, Get anwer from GPU
    hipMemcpy(y, cuda_y, y_size, hipMemcpyDeviceToHost);
    // hipMemcpy(x, cuda_x, x_size, hipMemcpyDeviceToHost);
    printAll("y:", y_h, y_w, y);
    // printAll("x:", x_h, x_w, x);


    hipFree(cuda_y);
    hipFree(cuda_x);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    cout << endl << "Time(milli) = " << milliseconds << endl << endl;

    return true;
}

