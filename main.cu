
#include "process.cu"

extern "C" {
    bool cudaGrayScaleImage(double *y, int y_h, int y_w, double *x, int x_h, int x_w, int x_d) {
        return _cudaGrayScaleImage(y, y_h, y_w, x, x_h, x_w, x_d);
    }
    bool grayScaleImage(double *y, int y_h, int y_w, double *x, int x_h, int x_w, int x_d) {
        cout << "Y(height: " << y_h << ", width: " << y_w << endl;
        cout << "X(height: " << x_h << ", width: " << x_w << ", depth: " << x_d << endl << endl;

        int d_w = 20;
        int d_h = 20;
        d_w = y_w;
        d_h = y_h;

        for (int i = 0; i < d_h; i++) {
            for (int j = 0; j < d_w; j++) {
                y[idx2(i, j, y_w)] = 0.2989 * x[idx3(i, j, 0, x_h, x_w)] + 0.5870 * x[idx3(i, j, 1, x_h, x_w)] + 0.1140 * x[idx3(i, j, 2, x_h, x_w)];
            }
        }

        return true;
    }
    int cppSum(double *a, int h, int w, int d) {
    
        double sum=0.0;
    
        for (int i = 0; i < h; i++)
        {
            for (int j = 0; j < w; j++)
            {
                for (int k = 0; k < d; k++)
                {
                    sum += 1; // a[i][j][k];
                }
            }
        }
        return sum;    
    }
}