
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
using namespace std;


void print(string prefix, int h, int w, double *A, int row, int cols) {

  cout << "\t" << prefix << "      ";

  for (int i = w*row; i < cols+w*row; i++)
    printf("(%d, %s%f),      ", i, (A[i] >= 0) ? "+" : "", A[i] );

  cout << "|      ";

  for (int i = w*row+w-cols; i < w*row+w; i++)
    printf("(%d, %s%f),      ", i, (A[i] >= 0) ? "+" : "", A[i] );

  cout << endl;
}

void printAll(string prefix, int h, int w, double *A) {
  int show = 2;
  int cols = 2;

  for (int i = 0; i < show; i++) {
    print(prefix, h, w, A, i, cols);
  }

  string line(cols*50, '_');
  cout << "                " << line << "\n";

  for (int i = h-show; i < h; i++) {
    print(prefix, h, w, A, i, cols);
  }
  cout << endl;
}

__device__ void insertion_sort(int n, int A[]) {
  for (int i = 1; i < n; i++) {
      int key_item = A[i];
      int j = i - 1;
      while (j >= 0 && A[j] > key_item) {
          A[j + 1] = A[j];
          j -= 1;
      }
      A[j + 1] = key_item;
  }
}

/**
 * @brief  Finding the Median of the 4 surrounding values and current value
 *
 * @param C Current
 * @param N North
 * @param E East
 * @param S South
 * @param W West
 * @return int for current location
 */
__device__ int median(int C, int N, int E, int S, int W) {
  int arr[] = {C, N, E, S, W};
  insertion_sort(5, arr);
  return arr[2];
}


/**
 * This is the implementation not using __shared__ memory, just accessing id - n and id + n
 */
__device__ void update(int n, int idx, int *Z, int *A)
{
  int x = idx % n;
  int y = idx / n;

  if (idx >= n*n) {
    // printf("idx(%d) >= n*n at: (x: %d, y: %d)\n", idx, x, y);
    return;
  }

  if ((y == 0) || (y == (n-1)) || (x == 0) || (x == (n-1))) {
    Z[idx] = A[idx];
  } else {
    Z[idx] = median(
        A[idx],
        A[idx - n],
        A[idx + 1],
        A[idx + n],
        A[idx - 1]
      );
  }
}

__device__ void getConstraints(int height, int width, int id, int threads, int bid, int blockStart, int blockSize, int* start, int* stop) {
  //int start, stop;

  int chunkSize = (height*width / threads); // 1000000/8195 = 122
  int remaining_nodes = height*width % threads;  // 576

  if (id < remaining_nodes) {
    chunkSize += 1;
    *start = chunkSize * id;  // 0, 123
    *stop = chunkSize * (id + 1);  // 123, 246

  } else {
    *start = remaining_nodes + chunkSize * id; // (576 + 122 * 577)
    *stop = remaining_nodes + chunkSize * (id + 1);

  }


}

// __global__ void iterations(int n, int threads, int *Z, int *A) {
//   int id = blockIdx.x * blockDim.x + threadIdx.x;
//   int blockStart = blockIdx.x * blockDim.x;

//   //if (id ==0) {
//   //  printf("Helllllloooooo");
//   //}
//   int start;
//   int stop;
//   getConstraints(n, id, threads, blockIdx.x, blockStart,  blockDim.x, &start, &stop);
//   // printf("ID: %d, blockStart: %d, Start: %d, Stop: %d\n", id, blockStart, start, stop);

//   for (int i = start; i < stop; i++) {
//     update(n, i, Z, A);
//   }

// }


/*
  Or Job (GPU):
    5 MINS:   salloc --account=eecs587f21_class --nodes=1 --gres=gpu --partition=gpu --time=00:05:00 --mem-per-cpu=5g
    1 HOUR:   salloc --account=engin1 --nodes=1 --gres=gpu --partition=gpu --mem-per-cpu=5g
  
  :
    module load cuda

  Running:
    nvcc -arch=sm_70 -std=c++11 main.cu && ./a.out <int:n>
*/


/**
 * @brief nvcc -arch=sm_70 -std=c++11 main.cu && ./a.out <int:n>
 *
 * @param argc
 * @param argv
 * @return int
 */
// int main(int argc, char const *argv[]) {




int idx3(int i, int j, int k, int h, int w) {
    return (i*w + j) * 3 + k;
    return i*h*w + j*w + k;
}
int idx2(int i, int j, int w) {
    return i*w + j;
}
int idx1(int i) {
    return i;
}

// void printMatrix3(numpyArray<double> a)
// {
//     Ndarray<double, 3> arr(a);

//     int rows = 5;
//     int cols = 5;
//     int ks = 1;

// 	for (int i = 0; i < rows; i++)
// 	{	
// 		cout << "\t" << "[";
// 		for (int j = 0; j < cols; j++)
// 		{
//             for (int k = 0; k < ks; k++)
//             {
//                 cout << arr[i][j][k] << "\t";
//             }
// 		}
// 		cout << "]" << endl;
// 	}
// }

// void printMatrix2(numpyArray<double> a)
// {
//     Ndarray<double, 2> arr(a);

//     int rows = 5;
//     int cols = 5;

//     for (int i = 0; i < rows; i++)
//     {	
//       cout << "\t" << "[";
//       for (int j = 0; j < cols; j++)
//       {
//           cout << arr[i][j] << "\t";
//       }
//       cout << "]" << endl;
//     }
// }

// cout << "Y(" << i << ")" << ", X(" << j << "), x(" << idx3(i, j, 0, x_h, x_w) << "), y(" << idx2(i, j, y_w) << ") :: ";
// cout << "["; 
// cout << x[idx3(i, j, 0, x_h, x_w)] << ", ";
// cout << x[idx3(i, j, 1, x_h, x_w)] << ", ";
// cout << x[idx3(i, j, 2, x_h, x_w)] << "]";
// cout << endl;